#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "array.h"
#include <cstdio>
#include <cmath>
#include <fstream>
#include <vector>

Timer timer = Timer();
double time_split;

const int FLAG_OBSTACLE = 1 << 1;
const int FLAG_KEEP_VELOCITY = 1 << 2;

typedef struct {
    unsigned int mantissa : 23;
    unsigned int exponent : 8;
    unsigned int sign : 1;
} floatparts;

const size_t Q = 19;
typedef array<float, Q> cell_t;
typedef vec3<float> vec3f;

struct gpu_t {
    int device;
    size_t mainGlobalIndex;
    size_t mainLayers;
    cell_t *data1;
    cell_t *data2;
    size_t mainOffset;
    size_t bottomPaddingOffset;
};

vec3<size_t> size;

size_t cells;
size_t bytesPerLayer;
size_t elementsPerLayer;

std::vector<gpu_t> gpuStructs;

hipStream_t *streams;

__managed__ float deltaT = 0.001f;

__managed__ float tau = 0.00065;
__managed__ float cellwidth = .01f;

bool desiredFanStatus = false;

bool pause = false;

__constant__ const array<vec3f, Q> offsets {
        0, 0, 0,   // 0
        -1, 0, 0,  // 1
        1, 0, 0,   // 2
        0, -1, 0,  // 3
        0, 1, 0,   // 4
        0, 0, -1,  // 5
        0, 0, 1,   // 6
        -1, -1, 0, // 7
        -1, 1, 0,  // 8
        1, -1, 0,  // 9
        1, 1, 0,   // 10
        -1, 0, -1, // 11
        -1, 0, 1,  // 12
        1, 0, -1,  // 13
        1, 0, 1,   // 14
        0, -1, -1, // 15
        0, -1, 1,  // 16
        0, 1, -1,  // 17
        0, 1, 1,   // 18
};

__constant__ const array<unsigned char, Q> opposite = {
        0,
        2, 1, 4, 3, 6, 5,
        10, 9, 8, 7, 14, 13, 12, 11, 18, 17, 16, 15
};

__constant__ const array<float, Q> wis {
    1.f / 3,
    1.f / 18,
    1.f / 18,
    1.f / 18,
    1.f / 18,
    1.f / 18,
    1.f / 18,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
    1.f / 36,
};

cell_t *u1;
cell_t *u2;

__device__ __host__ inline size_t pack(size_t w, size_t h, size_t d, size_t x, size_t y, size_t z) {
    return (z * h + y) * w + x;
}

__device__ __host__ inline float feq(size_t i, float p, const vec3f& v) {
    float wi = wis[i];
    float c = cellwidth;
    float dot = offsets[i] * c * v;
    return wi * p * (1 + (1 / (c * c)) * (3 * dot + (9 / (2 * c * c)) * dot * dot - (3.f / 2) * (v * v)));
}

__device__ inline void collisionStep(cell_t &cell) {
    float p = 0;
    float c = cellwidth;
    floatparts* parts = (floatparts*) &cell[0];
    if (parts->exponent == 255) {
        if (parts->mantissa & FLAG_OBSTACLE) {
            cell_t cell2 = cell;
            for (size_t i = 1; i < Q; i++) {
                cell[i] = cell2[opposite[i]];
            }
        }
        return;
    }
    vec3f vp {0, 0, 0};
    for (size_t i = 0; i < Q; i++) {
        p += cell[i];
        vp += offsets[i] * c * cell[i];
    }
    vec3f v = p == 0 ? vp : vp * (1 / p);

    for (size_t i = 0; i < Q; i++) {
        cell[i] = cell[i] + deltaT / tau * (feq(i, p, v) - cell[i]);
    }
}

__global__ void update(cell_t *dst, cell_t *src, size_t worksize, vec3<size_t> globalsize, size_t zoffset) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= worksize) {
        return;
    }
    size_t x = i % globalsize.x;
    size_t y = (i / globalsize.x) % globalsize.y;
    size_t z = (i / (globalsize.x * globalsize.y)) + zoffset;
    size_t index = i + zoffset * globalsize.x * globalsize.y;

    floatparts* parts = (floatparts*) &src[index][0];

    if (parts->exponent == 255) {
        if (parts->mantissa & FLAG_KEEP_VELOCITY) {
            dst[index] = src[index];
            return;
        }
    }

    for (int i = 0; i < Q; i++) {
        int sx = x + (int) offsets[i].x;
        int sy = y + (int) offsets[i].y;
        int sz = z + (int) offsets[i].z;
        if (sx < 0 || sy < 0 || sz < 0 || sx >= globalsize.x || sy >= globalsize.y || sz >= globalsize.z) {
            continue;
        }
        dst[index][i] = src[pack(globalsize.x, globalsize.y, globalsize.z, sx, sy, sz)][i];
    }

    collisionStep(dst[index]);
}

__device__ unsigned char floatToChar(float f) {
    return (unsigned char) min(max((f * 200.f + 1.f) * 127.f, 0.f), 255.f);
}

void syncStreams() {
    for (auto gpu : gpuStructs) {
        gpuErrchk(hipStreamSynchronize(streams[gpu.device]));
    }
}

__global__ void renderToBuffer(uchar4 *destImg, cell_t *srcU, vec3<size_t> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t z = 8;

    size_t iP = pack(size.x, size.y, size.z, x, y, z);
    size_t iI = (size.y - y - 1) * size.x + x; // Invert opengl image.
    vec3f p{};
    cell_t cell = srcU[iP];
    for (int i = 0; i < Q; i++) {
        p += offsets[i] * cell[i];
    }
    destImg[iI] = {
            floatToChar(p.x), floatToChar(p.y), floatToChar(p.z), 255
    };

}

void render(uchar4 *img, const int width, const int height) {
    simulateStep();
    dim3 threadsPerBlock(1, 1);
    dim3 numBlocks(size.x, size.y);
    renderToBuffer<<<numBlocks, threadsPerBlock>>>(img, gpuStructs[0].data1, size);
    hipDeviceSynchronize();
}

void initSimulation(size_t xdim, size_t ydim, size_t zdim, size_t gpus, const std::string &importFile) {
    size = {xdim, ydim, zdim};
    cells = xdim * ydim * zdim;

    size_t layersPerGpu = zdim / gpus;
    size_t remainder = zdim - layersPerGpu * gpus;
    elementsPerLayer = xdim * ydim;
    bytesPerLayer = elementsPerLayer * sizeof(cell_t);

    u1 = new cell_t[cells];
    u2 = new cell_t[cells];

    streams = new hipStream_t[gpus];

    size_t currentLayer = 0;
    gpuStructs = std::vector<gpu_t>();
    gpuStructs.reserve(gpus);

    for (int i = 0; i < gpus; i++) {
        gpuErrchk(hipSetDevice(i));
        gpuErrchk(hipStreamCreate(&streams[i]));
        gpu_t gpu{};
        gpu.device = i;
        int toppaddinglayers = i > 0 ? 1 : 0;
        int bottompaddinglayers = i < gpus - 1 ? 1 : 0;
        gpu.mainGlobalIndex = currentLayer * elementsPerLayer;
        gpu.mainLayers = layersPerGpu + (i < remainder ? 1 : 0);
        gpu.mainOffset = toppaddinglayers * elementsPerLayer;
        gpu.bottomPaddingOffset = gpu.mainOffset + gpu.mainLayers * elementsPerLayer;

        currentLayer += gpu.mainLayers;

        gpuErrchk(hipMalloc(&gpu.data1, (gpu.mainLayers + toppaddinglayers + bottompaddinglayers) * bytesPerLayer));
        gpuErrchk(hipMalloc(&gpu.data2, (gpu.mainLayers + toppaddinglayers + bottompaddinglayers) * bytesPerLayer));
        gpuStructs.push_back(gpu);
    }

    if (!importFile.empty()) {
        importFrame(importFile);
    } else {
        for (int x = 0; x < size.x; x++) {
            for (int y = 0; y < size.y; y++) {
                for (int z = 0; z < size.z; z++) {
                    for (int i = 0; i < Q; i++) {
                        float f = feq(i, 0.1f, {.001f, 0, 0});
                        u1[pack(size.x, size.y, size.z, x, y, z)][i] = f;
                    }

                    if (x <= 1 || y <= 1 || z <= 1 || x >= size.x - 2 || y >= size.y - 2 || z >= size.z - 2 || // x == 50 && (y >= 40 && y <= 45 || y >= 55 && y <= 60)) {
                        std::pow(x - 50, 2) + std::pow(y - 50, 2) + std::pow(z - 8, 2) <= 225) {
                        auto* parts = (floatparts*) &u1[pack(size.x, size.y, size.z, x, y, z)][0];
                        parts->sign = 0;
                        parts->exponent = 255;
                        if (x <= 1 || x >= size.x - 2 || y <= 1 || y >= size.y - 2 || z <= 1 || z >= size.z - 2) {
                            parts->mantissa = 1 << 22 | FLAG_KEEP_VELOCITY;
                        } else {
                            parts->mantissa = 1 << 22 | FLAG_OBSTACLE;
                        }
                    }
                }
            }
        }
    }

    for (auto gpu : gpuStructs) {
        gpuErrchk(hipMemcpyAsync(&gpu.data1[gpu.mainOffset], &u1[gpu.mainGlobalIndex], gpu.mainLayers * bytesPerLayer, hipMemcpyDefault, streams[gpu.device]));
    }
    syncStreams();
}

void togglePause() {
    pause = !pause;
}

void setFan(bool fan) {
    desiredFanStatus = fan;
}

bool getFan() {
    return desiredFanStatus;
}

void updateHost() {
    for (auto gpu : gpuStructs) {
        gpuErrchk(hipMemcpy(&u1[gpu.mainGlobalIndex], &gpu.data1[gpu.mainOffset], gpu.mainLayers * bytesPerLayer,
                             hipMemcpyDeviceToHost));
    }
}

void updateDevice() {
    for (auto gpu: gpuStructs) {
        gpuErrchk(hipMemcpy(&gpu.data1[gpu.mainOffset], &u1[gpu.mainGlobalIndex], gpu.mainLayers * bytesPerLayer,
                hipMemcpyHostToDevice));
    }
}


void simulateStep() {
    if (pause) {
        return;
    }

    for (int i = 1; i < gpuStructs.size(); i++) {
        // Copy bottom padding for i - 1
        gpuErrchk(hipMemcpyAsync(
                &gpuStructs[i - 1].data1[gpuStructs[i - 1].bottomPaddingOffset],
                &gpuStructs[i].data1[gpuStructs[i].mainOffset],
                bytesPerLayer, hipMemcpyDefault, streams[i - 1]
        ));
        // Copy top padding for i
        gpuErrchk(hipMemcpyAsync(
                gpuStructs[i].data1,
                &gpuStructs[i - 1].data1[gpuStructs[i - 1].bottomPaddingOffset - elementsPerLayer],
                bytesPerLayer, hipMemcpyDefault, streams[i]
        ));
    }

    syncStreams();
    time_split = timer.get();
    for (auto &gpu : gpuStructs) {
        hipSetDevice(gpu.device);
        dim3 threadsPerBlock(512);
        size_t worksize = size.x * size.y * gpu.mainLayers;
        dim3 numBlocks(
                (worksize + threadsPerBlock.x - 1) / threadsPerBlock.x
        );
        update<<<numBlocks, threadsPerBlock, 0, streams[gpu.device]>>>(
                gpu.data2, gpu.data1, worksize, size, gpu.mainOffset / elementsPerLayer
        );
        std::swap(gpu.data1, gpu.data2); // data1 is always pointing to up-to-date buffer.
    }
    std::swap(u1, u2);
    syncStreams();
}

void printLayer(size_t z) {
    gpuErrchk(hipMemcpy(u1, gpuStructs[0].data1, sizeof(cell_t) * cells, hipMemcpyDeviceToHost));

    for (size_t y = 0; y < 5u; y++) {
        for (size_t x = 0; x < 5u; x++) {
            cell_t v = u1[pack(size.x, size.y, size.z, x, y, z)];
            printf("(%f,%f,%f), ", v[0], v[1], v[2]);
        }
        printf("\n");
    }
    printf("\n");

}

void exportFrame(const std::string& filename) {

    updateHost();

    std::ofstream out;
    out.open(filename, std::ios::out | std::ios::binary);

    for (auto gpu : gpuStructs) {
        gpuErrchk(hipMemcpy(&u1[gpu.mainGlobalIndex], &gpu.data1[gpu.mainOffset], gpu.mainLayers * bytesPerLayer, hipMemcpyDeviceToHost));
    }

    out.write(reinterpret_cast<const char *>(u1), cells * sizeof(cell_t));

    out.close();
}

void importFrame(const std::string& importFile) {
    std::ifstream infile(importFile, std::ios_base::binary);
    infile.read(reinterpret_cast<char *>(u1), cells * sizeof(cell_t));
    updateDevice();
}

