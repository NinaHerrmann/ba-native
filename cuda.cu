#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <cstdio>
#include <cmath>

const glm::vec<3, size_t> SIZE(50, 50, 16);

const size_t CELLS = SIZE.x * SIZE.y * SIZE.z;

const float deltaT = 0.005f;

const float viscosity = 0.005;
const float cellwidth = 0.05;

const float EPSILON = 0.0001;

bool fanStatus = false;
bool desiredFanStatus = false;

__managed__ float currentTime;

glm::vec3 *u1;
glm::vec3 *u2;

glm::vec3 *cudau1;
glm::vec3 *cudau2;

float *p1;
float *p2;

float *cudap1;
float *cudap2;

__device__ __host__ inline size_t pack(size_t w, size_t h, size_t d, size_t x, size_t y, size_t z) {
    return (z * h + y) * w + x;
}

__global__ void updateU(glm::vec3 *src, glm::vec3 *dest, float *srcP, glm::vec<3, int> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    size_t z = blockIdx.z * blockDim.z + threadIdx.z + 1;

    size_t i = pack(size.x, size.y, size.z, x, y, z);
    size_t i1n = pack(size.x, size.y, size.z, x - 1, y, z);
    size_t i1p = pack(size.x, size.y, size.z, x + 1, y, z);
    size_t i2n = pack(size.x, size.y, size.z, x, y - 1, z);
    size_t i2p = pack(size.x, size.y, size.z, x, y + 1, z);
    size_t i3n = pack(size.x, size.y, size.z, x, y, z - 1);
    size_t i3p = pack(size.x, size.y, size.z, x, y, z + 1);
    glm::vec3 u = src[i];
    glm::vec3 u1n = src[i1n];
    glm::vec3 u1p = src[i1p];
    glm::vec3 u2n = src[i2n];
    glm::vec3 u2p = src[i2p];
    glm::vec3 u3n = src[i3n];
    glm::vec3 u3p = src[i3p];

    dest[i] = src[i] + deltaT * (viscosity / (cellwidth * cellwidth) * (u1n + u1p + u2n + u2p + u3n + u3p - 6.f * u)
            - 1.f / (2 * cellwidth) * (u1p.x - u1n.x + u2p.y - u2n.y + u3p.z - u3n.z) * u
                    - glm::vec3(srcP[i1n] - srcP[i1p], srcP[i2n] - srcP[i2p], srcP[i3n] - srcP[i3p]) / (2 * cellwidth));
}

__global__ void updatePSingleIteration(glm::vec3 *srcU, const float *srcP, float *destP, glm::vec<3, int> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    size_t z = blockIdx.z * blockDim.z + threadIdx.z + 1;
    size_t i = pack(size.x, size.y, size.z, x, y, z);
    size_t i1n = pack(size.x, size.y, size.z, x - 1, y, z);
    size_t i1p = pack(size.x, size.y, size.z, x + 1, y, z);
    size_t i2n = pack(size.x, size.y, size.z, x, y - 1, z);
    size_t i2p = pack(size.x, size.y, size.z, x, y + 1, z);
    size_t i3n = pack(size.x, size.y, size.z, x, y, z - 1);
    size_t i3p = pack(size.x, size.y, size.z, x, y, z + 1);

    float ud = (cellwidth / 2.f) * (
            srcU[i1p].x - srcU[i1n].x
            + srcU[i2p].y - srcU[i2p].y
            + srcU[i3p].z - srcU[i3p].z
    );

    destP[i] = ((cellwidth * cellwidth) / 6.f) * (srcP[i1p] + srcP[i1n] + srcP[i2p] + srcP[i2n] + srcP[i3p] + srcP[i3n] - ud);
}

__global__ void updateP(glm::vec3 *srcU, float *srcP, glm::vec<3, int> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    size_t z = blockIdx.z * blockDim.z + threadIdx.z + 1;
    size_t i = pack(size.x, size.y, size.z, x, y, z);
    size_t i1n = pack(size.x, size.y, size.z, x - 1, y, z);
    size_t i1p = pack(size.x, size.y, size.z, x + 1, y, z);
    size_t i2n = pack(size.x, size.y, size.z, x, y - 1, z);
    size_t i2p = pack(size.x, size.y, size.z, x, y + 1, z);
    size_t i3n = pack(size.x, size.y, size.z, x, y, z - 1);
    size_t i3p = pack(size.x, size.y, size.z, x, y, z + 1);

    float ud = (cellwidth / 2.f) * (
            srcU[i1p].x - srcU[i1n].x
            + srcU[i2p].y - srcU[i2p].y
            + srcU[i3p].z - srcU[i3p].z
    );

    float oldP = INFINITY;
    float p = srcP[i];
    do {
        oldP = p;
        p = ((cellwidth * cellwidth) / 6.f) * (srcP[i1p] + srcP[i1n] + srcP[i2p] + srcP[i2n] + srcP[i3p] + srcP[i3n] - ud);
        __syncthreads();
        srcP[i] = p;
        __syncthreads();
    } while((abs(p) - abs(oldP)) / (abs(p) + abs(oldP)) > EPSILON);
}

__global__ void updateUFromP(glm::vec3 *destU, const float *srcP, glm::vec<3, int> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    size_t z = blockIdx.z * blockDim.z + threadIdx.z + 1;
    size_t i = pack(size.x, size.y, size.z, x, y, z);
    size_t i1n = pack(size.x, size.y, size.z, x - 1, y, z);
    size_t i1p = pack(size.x, size.y, size.z, x + 1, y, z);
    size_t i2n = pack(size.x, size.y, size.z, x, y - 1, z);
    size_t i2p = pack(size.x, size.y, size.z, x, y + 1, z);
    size_t i3n = pack(size.x, size.y, size.z, x, y, z - 1);
    size_t i3p = pack(size.x, size.y, size.z, x, y, z + 1);

    destU[i] -= glm::vec3(srcP[i1p] - srcP[i1n], srcP[i2p] - srcP[i2n], srcP[i3p] - srcP[i3n]) / (2 * cellwidth);
}

__device__ unsigned char floatToChar(float f) {
    return (unsigned char) min(max((f + 1) * 127.f, 0.f), 255.f);
}

__global__ void renderToBuffer(uchar4 *destImg, glm::vec3 *srcU, glm::vec<3, int> size) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x; // Not calculating border cells.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t z = 8;

    size_t iP = pack(size.x, size.y, size.z, x, y, z);
    size_t iI = y * size.x + x;
    glm::vec3 p = srcU[iP];
    destImg[iI] = {
            floatToChar(p.x), floatToChar(p.y), floatToChar(p.z), 255
    };

}

void render(uchar4 *img, const int width, const int height) {
    simulateStep();
    dim3 threadsPerBlock(1, 1);
    dim3 numBlocks(SIZE.x, SIZE.y);
    renderToBuffer<<<numBlocks, threadsPerBlock>>>(img, cudau1, SIZE);
    hipDeviceSynchronize();
}

void setTime(float _time) {
    currentTime = _time;
}

void initSimulation() {
    u1 = new glm::vec3[CELLS];
    u2 = new glm::vec3[CELLS];

    gpuErrchk(hipMalloc(&cudau1, sizeof(glm::vec3) * CELLS));
    gpuErrchk(hipMalloc(&cudau2, sizeof(glm::vec3) * CELLS));

    p1 = new float[CELLS];
    p2 = new float[CELLS];
    gpuErrchk(hipMalloc(&cudap1, sizeof(float) * CELLS));
    gpuErrchk(hipMalloc(&cudap2, sizeof(float) * CELLS));
}

void turnOnFan() {
    gpuErrchk(hipMemcpy(u1, cudau1, sizeof(glm::vec3) * CELLS, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(u2, cudau2, sizeof(glm::vec3) * CELLS, hipMemcpyDeviceToHost));

    for (size_t i = 6; i < 10; i++) {
        for (size_t z = 0; z < SIZE.z; z++)  {
            u1[pack(SIZE.x, SIZE.y, SIZE.z, 0, i, z)] = glm::vec3(1.f, 0, 0);
            u2[pack(SIZE.x, SIZE.y, SIZE.z, 0, i, z)] = glm::vec3(1.f, 0, 0);
            u1[pack(SIZE.x, SIZE.y, SIZE.z, i, 0, z)] = glm::vec3(0, .75f, 0);
            u2[pack(SIZE.x, SIZE.y, SIZE.z, i, 0, z)] = glm::vec3(0, .75f, 0);
        }
    }

    gpuErrchk(hipMemcpy(cudau1, u1, sizeof(glm::vec3) * CELLS, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudau2, u2, sizeof(glm::vec3) * CELLS, hipMemcpyHostToDevice));
}

void turnOffFan() {
    gpuErrchk(hipMemcpy(u1, cudau1, sizeof(glm::vec3) * CELLS, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(u2, cudau2, sizeof(glm::vec3) * CELLS, hipMemcpyDeviceToHost));

    for (size_t i = 4; i < 12; i++) {
        for (size_t z = 0; z < SIZE.z; z++)  {
            u1[pack(SIZE.x, SIZE.y, SIZE.z, 0, i, z)] = glm::vec3(0, 0, 0);
            u2[pack(SIZE.x, SIZE.y, SIZE.z, 0, i, z)] = glm::vec3(0, 0, 0);
            u1[pack(SIZE.x, SIZE.y, SIZE.z, i, 0, z)] = glm::vec3(0, 0, 0);
            u2[pack(SIZE.x, SIZE.y, SIZE.z, i, 0, z)] = glm::vec3(0, 0, 0);
        }
    }

    gpuErrchk(hipMemcpy(cudau1, u1, sizeof(glm::vec3) * CELLS, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudau2, u2, sizeof(glm::vec3) * CELLS, hipMemcpyHostToDevice));
}

void setFan(bool fan) {
    desiredFanStatus = fan;
}

bool getFan() {
    return desiredFanStatus;
}

void simulateStep() {
    if (desiredFanStatus != fanStatus) {
        fanStatus = desiredFanStatus;
        if (fanStatus) {
            turnOnFan();
        } else {
            turnOffFan();
        }
    }
    dim3 threadsPerBlock(1, 1, 1);
    dim3 numBlocks(SIZE.x - 2, SIZE.y - 2, SIZE.z - 2);
    updateU<<<numBlocks, threadsPerBlock>>>(cudau1, cudau2, cudap1, SIZE);
    gpuErrchk(hipDeviceSynchronize());
    std::swap(u1, u2);
    std::swap(cudau1, cudau2);
    updateP<<<numBlocks, threadsPerBlock>>>(cudau1, cudap1, SIZE);
    /*for (int i = 0; i < 100; i++) {
        updatePSingleIteration<<<numBlocks, threadsPerBlock>>>(cudau1, cudap1, cudap2, glm::vec3(SIZE, SIZE, SIZE));
        gpuErrchk(hipDeviceSynchronize());
        std::swap(cudap1, cudap2);
        std::swap(p1, p2);
    }*/
    updateUFromP<<<numBlocks, threadsPerBlock>>>(cudau1, cudap1, SIZE);
    // printLayer(1);
}

void printP(size_t z) {
    gpuErrchk(hipMemcpy(p1, cudap1, sizeof(float) * CELLS, hipMemcpyDeviceToHost));
    for (size_t y = 0; y < SIZE.y; y++) {
        for (size_t x = 0; x < SIZE.x; x++) {
            printf("%f, ", p1[pack(SIZE.x, SIZE.y, SIZE.z, x, y, z)]);
        }
        printf("\n");
    }
    printf("\n");
}

void printLayer(size_t z) {
    gpuErrchk(hipMemcpy(u1, cudau1, sizeof(glm::vec3) * CELLS, hipMemcpyDeviceToHost));

    for (size_t y = 0; y < SIZE.y; y++) {
        for (size_t x = 0; x < SIZE.x; x++) {
            glm::vec3 v = u1[pack(SIZE.x, SIZE.y, SIZE.z, x, y, z)];
            printf("(%f,%f,%f), ", v.x, v.y, v.z);
        }
        printf("\n");
    }
    printf("\n");

}